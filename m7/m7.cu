/*
 * m7 algorithm
 *
 */


extern "C"
{
#include "sph/sph_sha2.h"
#include "sph/sph_keccak.h"
#include "sph/sph_ripemd.h"
#include "sph/sph_haval.h"
#include "sph/sph_tiger.h"
#include "sph/sph_whirlpool.h"
#include "sph/sph_blake.h"
}
#include "miner.h"
#include "cuda_helper.h"

extern int shortdevice_map[MAX_GPUS];

static uint64_t *d_hash[MAX_GPUS];
static uint64_t *KeccakH[MAX_GPUS];
static uint64_t *Sha512H[MAX_GPUS];
static uint64_t *d_prod0[MAX_GPUS];
static uint64_t *d_prod1[MAX_GPUS];

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

extern uint32_t m7_sha256_cpu_hash_300(int thr_id, int threads, uint32_t startNounce, uint64_t *d_nonceVector, uint64_t *d_hash, int order);

extern void m7_sha256_setBlock_120(void *data,const void *ptarget);
extern void m7_sha256_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order);
extern void m7_sha256_cpu_init(int thr_id, int threads);

extern void m7_sha512_cpu_init(int thr_id, int threads);
extern void m7_sha512_setBlock_120(void *pdata);
extern void m7_sha512_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void m7_ripemd160_cpu_init(int thr_id, int threads);
extern void m7_ripemd160_setBlock_120(void *pdata);
extern void m7_ripemd160_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void tiger192_cpu_init(int thr_id, int threads);
extern void tiger192_setBlock_120(void *pdata);
extern void m7_tiger192_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void m7_bigmul_init(int thr_id, int threads);
extern void m7_bigmul_unroll1_cpu(int thr_id, int threads,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order);
extern void m7_bigmul_unroll2_cpu(int thr_id, int threads,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order);

extern void cpu_mul(int thr_id, int threads, uint32_t alegs, uint32_t blegs, uint64_t *g_a, uint64_t *g_b, uint64_t *g_p, int order);
extern void cpu_mulT4(int thr_id, int threads, uint32_t alegs, uint32_t blegs, uint64_t *g_a, uint64_t *g_b, uint64_t *g_p, int order);
extern void mul_init();

extern void m7_keccak512_setBlock_120(void *pdata);
extern void m7_keccak512_cpu_hash(int thr_id, int threads, uint32_t startNounce, uint64_t *d_hash, int order);
extern void m7_keccak512_cpu_init(int thr_id, int threads);

extern void m7_whirlpool512_cpu_init(int thr_id, int threads, int flag);
extern void m7_whirlpool512_setBlock_120(void *pdata);
extern void m7_whirlpool512_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order);

extern void haval256_setBlock_120(void *data);
extern void m7_haval256_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order);

float tp_coef_f[8] = {1, 1, 1, 1, 1, 1, 1, 1};
extern bool opt_benchmark;

extern "C" int scanhash_m7(int thr_id, uint32_t *pdata, const uint32_t *ptarget, uint32_t max_nonce, unsigned long  *hashes_done) {
	if (opt_benchmark) ((uint32_t*)ptarget)[7] = 0x0000ff;
	int throughput = 256 * 256 * 100;
	const uint32_t FirstNonce = pdata[29];
	static bool init[8] = {0,0,0,0,0,0,0,0};

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		hipMalloc(&d_prod0[thr_id],      35 *sizeof(uint64_t) * throughput*tp_coef_f[thr_id]);
		hipMalloc(&d_prod1[thr_id],      38 *sizeof(uint64_t) * throughput*tp_coef_f[thr_id]);
		hipMalloc(&KeccakH[thr_id],     8 *sizeof(uint64_t) * throughput*tp_coef_f[thr_id]);

		m7_sha256_cpu_init(thr_id, throughput*tp_coef_f[thr_id]);
		m7_sha512_cpu_init(thr_id, throughput*tp_coef_f[thr_id]);
		m7_keccak512_cpu_init(thr_id, throughput*tp_coef_f[thr_id]);
        tiger192_cpu_init(thr_id, throughput*tp_coef_f[thr_id]);
		m7_whirlpool512_cpu_init(thr_id, throughput*tp_coef_f[thr_id],0);
		m7_ripemd160_cpu_init(thr_id, throughput*tp_coef_f[thr_id]);
		m7_bigmul_init(thr_id, throughput*tp_coef_f[thr_id]);
		mul_init();
		init[thr_id] = true;
	}

	if (pdata[28] == 0) pdata[28] = rand();	// multi-rig solo workaround

	m7_whirlpool512_setBlock_120((void*)pdata);
    m7_sha256_setBlock_120((void*)pdata,ptarget);
	m7_sha512_setBlock_120((void*)pdata);
	haval256_setBlock_120((void*)pdata);
	m7_keccak512_setBlock_120((void*)pdata);
	m7_ripemd160_setBlock_120((void*)pdata);
	tiger192_setBlock_120((void*)pdata);

	do {
		int order = 0;

		m7_keccak512_cpu_hash(thr_id, throughput*tp_coef_f[thr_id], pdata[29], KeccakH[thr_id], order++);
		m7_sha512_cpu_hash_120(thr_id, throughput*tp_coef_f[thr_id], pdata[29], d_prod1[thr_id], order++);
        cpu_mulT4(0, throughput*tp_coef_f[thr_id], 8, 8, d_prod1[thr_id], KeccakH[thr_id], d_prod0[thr_id],order); //64

        m7_whirlpool512_cpu_hash_120(thr_id, throughput*tp_coef_f[thr_id], pdata[29], KeccakH[thr_id], order++);
		cpu_mulT4(0, throughput*tp_coef_f[thr_id],8, 16, KeccakH[thr_id], d_prod0[thr_id], d_prod1[thr_id],order); //128

		m7_sha256_cpu_hash_120(thr_id, throughput*tp_coef_f[thr_id], pdata[29], KeccakH[thr_id], order++);
		cpu_mulT4(0, throughput*tp_coef_f[thr_id], 4, 24, KeccakH[thr_id], d_prod1[thr_id], d_prod0[thr_id],order); //96

		m7_haval256_cpu_hash_120(thr_id, throughput*tp_coef_f[thr_id], pdata[29], KeccakH[thr_id], order++);
		cpu_mulT4(0, throughput*tp_coef_f[thr_id], 4, 28, KeccakH[thr_id], d_prod0[thr_id], d_prod1[thr_id],order);  //112

		m7_tiger192_cpu_hash_120(thr_id, throughput*tp_coef_f[thr_id], pdata[29], KeccakH[thr_id], order++);
		m7_bigmul_unroll1_cpu(thr_id, throughput*tp_coef_f[thr_id], KeccakH[thr_id], d_prod1[thr_id], d_prod0[thr_id],order);

		m7_ripemd160_cpu_hash_120(thr_id, throughput*tp_coef_f[thr_id], pdata[29], KeccakH[thr_id], order++);
		m7_bigmul_unroll2_cpu(thr_id, throughput*tp_coef_f[thr_id], KeccakH[thr_id], d_prod0[thr_id], d_prod1[thr_id],order);

		uint32_t foundNonce = m7_sha256_cpu_hash_300(thr_id, throughput*tp_coef_f[thr_id], pdata[29], NULL, d_prod1[thr_id], order);
		if  (foundNonce != 0xffffffff) {
			*hashes_done = pdata[29] - FirstNonce + throughput*tp_coef_f[thr_id];
            pdata[29] = foundNonce;
			return 1;
        }

		pdata[29] += throughput*tp_coef_f[thr_id];

	} while (((uint64_t)max_nonce > ((uint64_t)(pdata[29]) + (uint64_t)throughput*tp_coef_f[thr_id])) && !work_restart[thr_id].restart);

	*hashes_done = pdata[29] - FirstNonce;
	return 0;
}
